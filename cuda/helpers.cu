#include "hip/hip_runtime.h"
#include "helpers.h"
#include <unistd.h> 
#include <hip/hip_runtime.h>

void CUDA_CHECK(hipError_t hipError_t) {
    if (hipError_t != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }
}

void CHECK_CUBLAS(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS error: %d\n", status);
        exit(1);
    }
}

// Matrix functions

__global__ void elementwiseMult(int m, int n, double* matrix, double* matrix2){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * m + col;
    if (idx < m * n) {
        matrix[idx] *= matrix2[idx];
    }
}

__global__ void fill_A_B(int m, int n, double* matrix){
    // Fill matrix with values between A and B
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * m + col;
    if (idx < m * n) {
        matrix[idx] = matrix[idx] * 2 - 1;
    }
}

void print_matrix(double *mat, int m, int n){
    
    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
            printf("%f ", min(double(1000), max(double(-1000), mat[m * j + i])));
        }
        printf("\n");
    }
    printf("\n");
}

void print_device(double *dev, int m, int n){
    return;
    double *temp = (double *)calloc(sizeof(double), m * n);
    CUDA_CHECK(hipMemcpy(temp, dev, sizeof(double) * m * n, hipMemcpyDeviceToHost));
    print_matrix(temp, m, n);
    free(temp);
}

void createRandomMatrix(int m, int n, double* matrix) {
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(prng, 123);
    hiprandGenerateUniformDouble(prng, matrix, m * n);
    hiprandDestroyGenerator(prng);
    dim3 threads(16,16);
    dim3 blocks((m + threads.x - 1)  / threads.x, (n + threads.y - 1)  / threads.y);
    fill_A_B<<<blocks, threads>>>(m, n, matrix);
}
#include "hip/hip_runtime.h"
#include "helpers.h"

void CUDA_CHECK(hipError_t hipError_t) {
    if (hipError_t != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(hipError_t));
        exit(1);
    }
}

void CHECK_CUBLAS(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS error: %d\n", status);
        exit(1);
    }
}

// Matrix functions
__global__ void elementwiseMult(int m, int n, double* matrix, double* matrix2){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // if (idx < m * n && matrix[idx] < 0) {
    if (idx < m * n) {
        matrix[idx] *= matrix2[idx];
    }
}

__global__ void fill_A_B(int m, int n, double* matrix){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < m * n) {
        matrix[idx] = matrix[idx] * 2 - 1;
    }
}

void print_matrix(double *mat, int m, int n){
    for (int i = 0; i < m; i++){
        for (int j = 0; j < n; j++){
            printf("%f ", min(double(6900000), max(double(-6900000), mat[m * j + i])));
        }
        printf("\n");
    }
}

void print_device(double *dev, int m, int n){
    double *temp = (double *)calloc(sizeof(double), m * n);
    CUDA_CHECK(hipMemcpy(temp, dev, sizeof(double) * m * n, hipMemcpyDeviceToHost));
    print_matrix(temp, m, n);
    free(temp);
}

void createRandomMatrix(int m, int n, double* matrix) {
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(prng, 123);
    hiprandGenerateUniformDouble(prng, matrix, m * n);
    hiprandDestroyGenerator(prng);
    int nBlocks = (m * n)/BLOCK_SIZE;
    fill_A_B<<<512, BLOCK_SIZE>>>(m, n, matrix);
}